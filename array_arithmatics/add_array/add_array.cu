#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime> 

//Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
    {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// CPU version of VecAdd
void VecAddCPU(float* A, float* B, float* C, int N) {
    for (int i = 0; i < N; i++) {
        C[i] = A[i] + B[i];
    }
}


// Host code
int main()
{
    int N = 102400;
    size_t size = N * sizeof(float);
    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);
    float* h_C_CPU = (float*)malloc(size);  // For CPU result


    // ∕∕ Initialize input vectors
    for(int i=0;i<N;i++){
        h_A[i]=1.0 * i;
        h_B[i]=2.0 * i;
    }

    // CPU timing
    clock_t start_cpu = clock();
    VecAddCPU(h_A, h_B, h_C_CPU, N);
    clock_t end_cpu = clock();
    double cpu_time = double(end_cpu - start_cpu) / CLOCKS_PER_SEC;
    printf("Time taken by CPU: %f milliseconds\n", cpu_time*1000);

    // ∕∕ Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    // ∕∕ Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


    // Create CUDA events for timing GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // ∕∕ Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Start recording the GPU time
    hipEventRecord(start);
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    // Wait for the kernel to finish and get the time taken
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by GPU: %f milliseconds\n", milliseconds);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_CPU);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}